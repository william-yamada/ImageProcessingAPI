#include "hip/hip_runtime.h"
#include "image_processor.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void invert_kernel(unsigned char* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) data[idx] = 255 - data[idx];
}

bool cuda_available() {
    int count;
    hipGetDeviceCount(&count);
    return count > 0;
}

std::vector<unsigned char> CUDAImageProcessor::process(const std::vector<unsigned char>& image, int width, int height) {
    std::vector<unsigned char> output(image.size());
    unsigned char* d_image;
    hipMalloc(&d_image, image.size());
    hipMemcpy(d_image, image.data(), image.size(), hipMemcpyHostToDevice);
    invert_kernel<<<(image.size()+255)/256, 256>>>(d_image, image.size());
    hipMemcpy(output.data(), d_image, image.size(), hipMemcpyDeviceToHost);
    hipFree(d_image);
    return output;
}
